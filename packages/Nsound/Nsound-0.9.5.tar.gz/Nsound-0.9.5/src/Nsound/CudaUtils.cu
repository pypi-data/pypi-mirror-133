///////////////////////////////////////////////////////////////////////////////
//
//  $Id: CudaUtils.cu 699 2011-10-03 20:56:04Z weegreenblobbie $
//
//  Nsound is a C++ library and Python module for audio synthesis featuring
//  dynamic digital filters. Nsound lets you easily shape waveforms and write
//  to disk or plot them. Nsound aims to be as powerful as Csound but easy to
//  use.
//
//  Copyright (c) 2008 to Present Nick Hilton
//
//  weegreenblobbie2_gmail_com (replace '_' with '@' and '.')
//
///////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////
//
//  This program is free software; you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation; either version 2 of the License, or
//  (at your option) any later version.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU Library General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
//
///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <Nsound/CudaUtils.h>
#include <string.h>
#include <stdio.h>

// The nvcc compiler doesn't handle namespaces the same way GCC does, as of
// version: Cuda compilation tools, release 3.2, V0.2.1221

Nsound::Context::State
Nsound::Context::state_ = Nsound::Context::BOOTING_;

///////////////////////////////////////////////////////////////////////////////
void
Nsound::
Context::
init(int flags)
{
    if(state_ == BOOTING_)
    {
        if(hipSuccess == hipInit(flags))
        {
            state_ = INITALIZED_;
        }
        else
        {
            checkForCudaError();
            state_ = ERROR_;
        }
    }
}

// :mode=c++: jEdit modeline
