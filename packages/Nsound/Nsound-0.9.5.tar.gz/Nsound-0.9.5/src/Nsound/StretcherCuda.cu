#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
//  $Id$
//
//  Nsound is a C++ library and Python module for audio synthesis featuring
//  dynamic digital filters. Nsound lets you easily shape waveforms and write
//  to disk or plot them. Nsound aims to be as powerful as Csound but easy to
//  use.
//
//  Copyright (c) 2008 to Present Nick Hilton
//
//  weegreenblobbie2_gmail_com (replace '_' with '@' and '.')
//
///////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////
//
//  This program is free software; you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation; either version 2 of the License, or
//  (at your option) any later version.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU Library General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
//
///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <Nsound/CudaUtils.h>
#include <Nsound/StretcherCuda.h>

#include <stdio.h>
#include <string.h>

using namespace Nsound;

template<typename T>
__global__
void
calculateRSS(
    T *                rss,
    const T *          A,
    const T *          B,
    const unsigned int window_length,
    const unsigned int max_delta,
    const unsigned int thread_offset)
{
    const unsigned int offset = blockIdx.x + thread_offset;

    // Is this a valid thread?
    if(offset >= max_delta) return;

    T sum = 0.0;

    for(unsigned int i = 0; i < window_length; ++i)
    {
        T diff = A[i] - B[i+offset];

        sum += diff * diff;
    }

    rss[offset] = sum;
}


///////////////////////////////////////////////////////////////////////////////
//
// Calculate the residual sum square for all max_delta test cases in parallel
// on the cuda card.
void
Nsound::
searchForBestMatch_f64(
    double *           rss,
    const double *     source,
    const unsigned int source_index,
    const unsigned int search_index,
    const unsigned int window_length,
    const unsigned int max_delta)
{
    Nsound::Context context;
    context.init(0);

    if(context.state_ != Nsound::Context::INITALIZED_)
    {
        rss[0] = -1*context.state_;
        rss[1] = -1*context.state_;
        rss[2] = -1*context.state_;
        return;
    }

    // Copy vectors to device
    // Copy source & rss_array to device
    double * gpuA   = NULL;
    double * gpuB   = NULL;
    double * gpuRSS = NULL;

    const unsigned int ABSIZE = sizeof(double) * (max_delta + window_length);
    const unsigned int RSSIZE = sizeof(double) * (max_delta);

    // Allocate GPU memory
    hipMalloc((void**)&gpuA,   ABSIZE); checkForCudaError();
    hipMalloc((void**)&gpuB,   ABSIZE); checkForCudaError();
    hipMalloc((void**)&gpuRSS, RSSIZE); checkForCudaError();

    // Copy the data to the device
    hipMemcpy(
        gpuA,
        source + source_index,
        ABSIZE,
        hipMemcpyHostToDevice); checkForCudaError();

    hipMemcpy(
        gpuB,
        source + search_index,
        ABSIZE,
        hipMemcpyHostToDevice); checkForCudaError();

    unsigned int threads_per_block = 65536 / max_delta;

//~    for(
//~        unsigned int thread_offset = 0;
//~        thread_offset < max_delta;
//~        thread_offset += threads_per_block)
//~    {

        calculateRSS<double> <<<max_delta, 1>>>(
            gpuRSS,
            gpuA,
            gpuB,
            window_length,
            max_delta,
            0); checkForCudaError();

        // Copy the data back to the host
        hipMemcpy(
            rss,
            gpuRSS,
            RSSIZE,
            hipMemcpyDeviceToHost); checkForCudaError();
//~    }
}

void
Nsound::
searchForBestMatch_f32(
    float *           rss,
    const float *     source,
    const unsigned int source_index,
    const unsigned int search_index,
    const unsigned int window_length,
    const unsigned int max_delta)
{
    Nsound::Context context;
    context.init(0);

    if(context.state_ != Nsound::Context::INITALIZED_)
    {
        rss[0] = -1*context.state_;
        rss[1] = -1*context.state_;
        rss[2] = -1*context.state_;
        return;
    }

    // Copy vectors to device
    // Copy source & rss_array to device
    float * gpuA   = NULL;
    float * gpuB   = NULL;
    float * gpuRSS = NULL;

    const unsigned int ABSIZE = sizeof(float) * (max_delta + window_length);
    const unsigned int RSSIZE = sizeof(float) * (max_delta);

    // Allocate GPU memory
    hipMalloc((void**)&gpuA,   ABSIZE); checkForCudaError();
    hipMalloc((void**)&gpuB,   ABSIZE); checkForCudaError();
    hipMalloc((void**)&gpuRSS, RSSIZE); checkForCudaError();

    // Copy the data to the device
    hipMemcpy(
        gpuA,
        source + source_index,
        ABSIZE,
        hipMemcpyHostToDevice); checkForCudaError();

    hipMemcpy(
        gpuB,
        source + search_index,
        ABSIZE,
        hipMemcpyHostToDevice); checkForCudaError();

    // Magic Number, applies to nick's cuda card
    unsigned int threads_per_block = 512;

    for(
        unsigned int thread_offset = 0;
        thread_offset < max_delta;
        thread_offset += threads_per_block)
    {

        calculateRSS<float> <<<threads_per_block, 1>>>(
            gpuRSS,
            gpuA,
            gpuB,
            window_length,
            max_delta,
            thread_offset); checkForCudaError();
    }

    // Copy the data back to the host
    hipMemcpy(
        rss,
        gpuRSS,
        RSSIZE,
        hipMemcpyDeviceToHost); checkForCudaError();
}

// :mode=c++: jEdit modeline
